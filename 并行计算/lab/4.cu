#include <iostream>
#include <vector>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>

using namespace std;

// 函数原型：用于按列索引排序稀疏矩阵
void sortSparseMatrix(vector<int>& sparse_rows, vector<int>& sparse_cols, vector<int>& sparse_vals, vector<int>& idx);

__global__ void sparseDenseMatMult(int M, int N, int P, int K,
                                   int *dense, int *sparse_rows, int *sparse_col_ptr, int *sparse_col_idx, int *sparse_vals,
                                   int *result) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // 计算当前线程处理的行索引
    int col = blockIdx.x * blockDim.x + threadIdx.x; // 计算当前线程处理的列索引

    if (row < M && col < P) {
        int value = 0;
        int start = sparse_col_ptr[col];    // 当前列的起始索引
        int end = sparse_col_ptr[col + 1];  // 当前列的结束索引

        for (int k = start; k < end; ++k) {
            if (sparse_col_idx[k] == col) { // 检查当前非零元素是否在当前列
                int idx = sparse_rows[k];   // 获取稀疏矩阵行索引
                value += dense[row * N + idx] * sparse_vals[k]; // 计算乘积并累加到结果值
            }
        }
        result[row * P + col] = value; // 将结果存入结果矩阵中
    }
}

int main() {
    int M, N, P, K;
    cin >> M >> N >> P >> K; // 输入稠密矩阵的维度和稀疏矩阵的非零元素个数

    // 输入稠密矩阵
    vector<int> dense_matrix(M * N);
    for (int i = 0; i < M * N; ++i) {
        cin >> dense_matrix[i];
    }

    // 输入稀疏矩阵的非零元素
    vector<int> sparse_rows(K), sparse_cols(K);
    vector<int> sparse_vals(K);
    for (int i = 0; i < K; ++i) {
        cin >> sparse_rows[i] >> sparse_cols[i] >> sparse_vals[i];
    }

    // 调用排序函数对稀疏矩阵进行排序
    vector<int> idx(K);
    sortSparseMatrix(sparse_rows, sparse_cols, sparse_vals, idx);

    // 创建稀疏矩阵的压缩稀疏列（CSC）格式所需的 sparse_col_ptr 和 sparse_col_idx 数组
    vector<int> sparse_col_ptr(P + 1, 0);
    vector<int> sparse_col_idx(K);

    for (int k = 0; k < K; ++k) {
        sparse_col_ptr[sparse_cols[k] + 1]++;
        sparse_col_idx[k] = sparse_cols[k];
    }

    // 计算 sparse_col_ptr 数组中的偏移量
    for (int j = 1; j <= P; ++j) {
        sparse_col_ptr[j] += sparse_col_ptr[j - 1];
    }

    // 在设备上分配内存
    int *d_dense, *d_sparse_vals, *d_result;
    int *d_sparse_rows, *d_sparse_col_ptr, *d_sparse_col_idx;

    hipMalloc(&d_dense, M * N * sizeof(int));
    hipMalloc(&d_sparse_rows, K * sizeof(int));
    hipMalloc(&d_sparse_col_ptr, (P + 1) * sizeof(int));
    hipMalloc(&d_sparse_col_idx, K * sizeof(int));
    hipMalloc(&d_sparse_vals, K * sizeof(int));
    hipMalloc(&d_result, M * P * sizeof(int));

    // 将数据从主机内存复制到设备内存
    hipMemcpy(d_dense, dense_matrix.data(), M * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sparse_rows, sparse_rows.data(), K * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sparse_col_ptr, sparse_col_ptr.data(), (P + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sparse_col_idx, sparse_col_idx.data(), K * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sparse_vals, sparse_vals.data(), K * sizeof(int), hipMemcpyHostToDevice);

    // 定义 CUDA kernel 的线程块和网格
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((P + 15) / 16, (M + 15) / 16);

    // 调用 CUDA kernel
    sparseDenseMatMult<<<numBlocks, threadsPerBlock>>>(M, N, P, K, d_dense, d_sparse_rows, d_sparse_col_ptr, d_sparse_col_idx, d_sparse_vals, d_result);

    // 将结果从设备内存复制回主机内存
    vector<int> result_matrix(M * P);
    hipMemcpy(result_matrix.data(), d_result, M * P * sizeof(int), hipMemcpyDeviceToHost);

    // 输出结果矩阵
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < P; ++j) {
            cout << result_matrix[i * P + j] << " ";
        }
        cout << endl;
    }

    // 释放设备内存
    hipFree(d_dense);
    hipFree(d_sparse_rows);
    hipFree(d_sparse_col_ptr);
    hipFree(d_sparse_col_idx);
    hipFree(d_sparse_vals);
    hipFree(d_result);

    return 0;
}

// 定义排序稀疏矩阵的函数
void sortSparseMatrix(vector<int>& sparse_rows, vector<int>& sparse_cols, vector<int>& sparse_vals, vector<int>& idx) {
    iota(idx.begin(), idx.end(), 0); // 初始化索引数组
    sort(idx.begin(), idx.end(), [&sparse_cols](int i1, int i2) { return sparse_cols[i1] < sparse_cols[i2]; });

    vector<int> sorted_sparse_cols(sparse_cols.size());
    vector<int> sorted_sparse_rows(sparse_rows.size());
    vector<int> sorted_sparse_vals(sparse_vals.size());

    for (size_t i = 0; i < idx.size(); ++i) {
        sorted_sparse_cols[i] = sparse_cols[idx[i]];
        sorted_sparse_rows[i] = sparse_rows[idx[i]];
        sorted_sparse_vals[i] = sparse_vals[idx[i]];
    }

    sparse_cols = sorted_sparse_cols;
    sparse_rows = sorted_sparse_rows;
    sparse_vals = sorted_sparse_vals;
}
